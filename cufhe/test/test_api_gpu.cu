/**
 * Copyright 2018 Wei Dai <wdai3141@gmail.com>
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

// Include these two files for GPU computing.
#include <include/cufhe_gpu.cuh>
using namespace cufhe;

#include <iostream>
using namespace std;

#include <vector>

void NandCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = 1 - in0.message_ * in1.message_;
}

void OrCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) > 0;
}

void AndCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = in0.message_ * in1.message_;
}

void XorCheck(Ptxt& out, const Ptxt& in0, const Ptxt& in1) {
  out.message_ = (in0.message_ + in1.message_) & 0x1;
}

int main() {
  const int num_gpus = 2; // Set the number of GPUs you want to use
  const uint32_t kNumTestsPerGPU = 16; // Number of tests per GPU
  const uint32_t kNumLevels = 4;

  // Create a vector to hold devices and streams for each GPU
  std::vector<int> devices(num_gpus);
  std::vector<Stream*> streams(num_gpus);

  for (int i = 0; i < num_gpus; i++) {
    devices[i] = i;
    streams[i] = new Stream[num_gpus];
    for (int j = 0; j < num_gpus; j++) {
      streams[i][j].Create();
    }
  }

  SetSeed(); // set random seed

  PriKey pri_key; // private key
  PubKey pub_key; // public key
  Ptxt* pt = new Ptxt[2 * kNumTests];
  Ctxt* ct = new Ctxt[2 * kNumTests];
  Synchronize();
  bool correct;

  cout<< "------ Key Generation ------" <<endl;
  KeyGen(pub_key, pri_key);
  // Alternatively ...
  // PriKeyGen(pri_key);
  // PubKeyGen(pub_key, pri_key);

  cout<< "------ Test Encryption/Decryption ------" <<endl;
  cout<< "Number of tests:\t" << kNumTests <<endl;
  correct = true;
  for (int i = 0; i < kNumTests; i ++) {
    pt[i].message_ = rand() % Ptxt::kPtxtSpace;
    Encrypt(ct[i], pt[i], pri_key);
    Decrypt(pt[kNumTests + i], ct[i], pri_key);
    if (pt[kNumTests + i].message_ != pt[i].message_) {
      correct = false;
      break;
    }
  }
  if (correct)
    cout<< "PASS" <<endl;
  else
    cout<< "FAIL" <<endl;

  // Test NAND gate (and other gates) on multiple GPUs
  std::cout << "------ Test NAND Gate ------" << std::endl;
  std::cout << "Number of tests per GPU:\t" << kNumTestsPerGPU << std::endl;
  bool correct = true;

  for (int i = 0; i < num_gpus; i++) {
    hipSetDevice(devices[i]);
    Initialize(pub_key); // essential for GPU computing

    for (int j = 0; j < kNumTestsPerGPU; j++) {
      // Create and encrypt data here
      Ptxt* pt = new Ptxt[2 * kNumTestsPerGPU];
      Ctxt* ct = new Ctxt[2 * kNumTestsPerGPU];
      for (int k = 0; k < 2 * kNumTestsPerGPU; k++) {
        pt[k].message_ = rand() % Ptxt::kPtxtSpace;
        Encrypt(ct[k], pt[k], pri_key);
      }
      Synchronize();
      
      // Perform gate operations on multiple GPUs here
      for (int k = 0; k < kNumTestsPerGPU; k++) {
        Nand(ct[k], ct[k], ct[k + kNumTestsPerGPU], streams[i][j]);
        Or(ct[k], ct[k], ct[k + kNumTestsPerGPU], streams[i][j]);
        And(ct[k], ct[k], ct[k + kNumTestsPerGPU], streams[i][j]);
        Xor(ct[k], ct[k], ct[k + kNumTestsPerGPU], streams[i][j]);
      }

      Synchronize();
      
      // Decrypt and check results here
      int cnt_failures = 0;
      for (int k = 0; k < kNumTestsPerGPU; k++) {
        NandCheck(pt[k], pt[k], pt[k + kNumTestsPerGPU]);
        OrCheck(pt[k], pt[k], pt[k + kNumTestsPerGPU]);
        AndCheck(pt[k], pt[k], pt[k + kNumTestsPerGPU]);
        XorCheck(pt[k], pt[k], pt[k + kNumTestsPerGPU]);
        Decrypt(pt[k + kNumTestsPerGPU], ct[k], pri_key);
        if (pt[k + kNumTestsPerGPU].message_ != pt[k].message_) {
          correct = false;
          cnt_failures += 1;
        }
      }
      if (!correct) {
        std::cout << "GPU " << i << " Test " << j << " FAIL:\t" << cnt_failures << "/" << kNumTestsPerGPU << std::endl;
      }
      
      // Clean up resources for this GPU
      delete[] ct;
      delete[] pt;
    }
    
    CleanUp(); // essential to clean and deallocate data
  }

  // Clean up streams and devices
  for (int i = 0; i < num_gpus; i++) {
    hipSetDevice(devices[i]);
    for (int j = 0; j < num_gpus; j++) {
      streams[i][j].Destroy();
    }
    delete[] streams[i];
  }

  return 0;
}
